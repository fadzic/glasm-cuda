#include "hip/hip_runtime.h"
/*
 *  Copyright (C) 2014  Kristijan Lenac
 *
 *  This program is free software; you can redistribute it and/or modify
 *  it under the terms of the GNU General Public License as published by
 *  the Free Software Foundation; either version 2 of the License, or
 *  (at your option) any later version.
 *
 *  This program is distributed in the hope that it will be useful,
 *  but WITHOUT ANY WARRANTY; without even the implied warranty of
 *  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 *  GNU General Public License for more details.
 *
 *  You should have received a copy of the GNU General Public License along
 *  with this program; if not, write to the Free Software Foundation, Inc.,
 *  51 Franklin Street, Fifth Floor, Boston, MA 02110-1301 USA.
 */


//#include <iostream>
//using namespace std;


#include <stdlib.h>

#include "sga_cuda.h"
#include "utils.h"


//---------------------------------------------------------------------------
//------------------------ sga ----------------------------------------------
//---------------------------------------------------------------------------

namespace CUDA{

	// definizioni globali
	int nbitx;
	int nbity;
	int nbitrot;
	int popsize;
	int *d_popsize;
	int maxruns;
	int maxgen;
	double pcross;
	double pmutation;

	const int BITS_PER_BYTE = 8;// number of bits per byte on this machine
	//const int UINTSIZE = BITS_PER_BYTE*sizeof(unsigned int);// # of bits in unsigned

	int lchrom;
	int *d_lchrom;

	__device__ struct individual *oldpop;/* last generation of individuals */
	__device__ struct individual *newpop;/* next generation of individuals */
	__device__ struct bestever bestfit; /* fittest individual so far */
	__device__ double sumfitness;/* summed fitness for entire population */
	__device__ double mymax;/* maximum fitness of population */
	__device__ double avg;/* average fitness of population */
	__device__ double mymin;/* minumum fitness of population */

	int gen;/* current generation number */
	int *d_gen;
	int run;/* current run number */
	__device__ int nmutation;/* number of mutations */
	__device__ int ncross;/* number of crossovers */

	int *rand_numbers;
	int *d_rand_numbers;

	//ovo ce trebat promjenit. Najlakse je prepisat objfun na device
	void (*sga_objfun)(struct individual *critter);

	void set_sga_objfun(void (*objfun)(struct individual *critter))
	{
		sga_objfun=objfun;
	}
	//--------------------------------------------------------------

	void sga_parameters(int unbitx, int unbity, int unbitrot, int upopsize, int umaxruns, int umaxgen, double upcross, double upmutation)
	{
		nbitx=unbitx;
		nbity=unbity;
		nbitrot=unbitrot;
		lchrom=nbitx+nbity+nbitrot;

		popsize=upopsize;
		if((popsize%2) != 0) popsize++;
		maxruns=umaxruns;
		maxgen=umaxgen;
		pcross=upcross;
		pmutation=upmutation;

		hipMalloc((void **)&d_popsize,sizeof(int));
		hipMemcpy(d_popsize, &popsize, sizeof(int), hipMemcpyHostToDevice);

		hipMalloc((void **)&d_lchrom,sizeof(int));
		hipMemcpy(d_lchrom, &lchrom, sizeof(int), hipMemcpyHostToDevice);
	}

	__device__ static int *choices, nremain;
	__device__ static float *fraction;

	__device__ void d_selectMemory(int *popsize)
	{
		unsigned nbytes;

		nbytes = *popsize*sizeof(int);
		if((choices = (int *) malloc(nbytes)) == NULL) {} //Errore: Non posso allocare memoria dinamica per choices
		nbytes = *popsize*sizeof(float);
		if((fraction = (float *) malloc(nbytes)) == NULL) {} //Errore: Non posso allocare memoria dinamica per fraction
	}

	__global__ void d_initmalloc(int *popsize)
	{
		unsigned nbytes;

		nbytes = *popsize*sizeof(struct individual); // memory for old and new populations of individuals
		if((oldpop = (struct individual *) malloc(nbytes)) == NULL) {} //Errore: Non posso allocare memoria dinamica per oldpop
		if((newpop = (struct individual *) malloc(nbytes)) == NULL) {} //Errore: Non posso allocare memoria dinamica per newpop

		d_selectMemory(popsize);

		nmutation = 0;// initialize global counters/values
		ncross = 0;
		bestfit.fitness = 0.0;
		bestfit.generation = 0;
	}


	__device__ void d_sga_objfun(individual *critter)
	{
		critter->fitness = 4;
	}

	__global__ void d_initialize(int *lchrom, int* rand_num)
	{
		int j, k;// initialize population
		unsigned mask = 1;

		j = blockIdx.x;

		oldpop[j].chrom = 0;
		for(k = 0; k < *lchrom; k++)
		{
			oldpop[j].chrom = (oldpop[j].chrom<<1);
			if(rand_num[j]<499) oldpop[j].chrom = oldpop[j].chrom|mask;
		}
		oldpop[j].parent[0] = 0;// Initialize parent info
		oldpop[j].parent[1] = 0;
		oldpop[j].xsite = 0;

		d_sga_objfun(&(oldpop[j]));// Evaluate initial fitness
	}

	__global__ void d_statistics_1()
	{
		sumfitness = 0.0;
		mymin = oldpop[0].fitness;
		mymax = oldpop[0].fitness;
	}

	__global__ void d_statistics_2(int *gen)
	{
		struct individual *pop = oldpop;

		int j;
		j = blockIdx.x;

		__syncthreads();

		sumfitness = sumfitness + pop[j].fitness;// Accumulate
		if(pop[j].fitness > mymax) mymax = pop[j].fitness;// New maximum
		if(pop[j].fitness < mymin) mymin = pop[j].fitness;// New minimum
		if(pop[j].fitness > bestfit.fitness)// new global best-fit individual
		{
			bestfit.chrom = pop[j].chrom;
			bestfit.fitness= pop[j].fitness;
			bestfit.generation = *gen;
		}
	}

	__global__ void d_statistics_3(int *popsize)
	{
		avg = sumfitness/(*popsize);// Calculate average
	}

	void statistics_old()
	{
		d_statistics_1<<<1,1>>>();
		d_statistics_2<<<popsize,1>>>(d_gen);
		d_statistics_3<<<1,1>>>(d_popsize);
	}

	void randomize()
	{
		for(int i=0; i<popsize; i++)
		{
			rand_numbers[i] = _random(1000);
		}

		hipMemcpy(d_rand_numbers, rand_numbers, sizeof(int), hipMemcpyHostToDevice);
	}

	void initialize()
	{
		randomize();
		d_initialize<<<popsize,1>>>(d_lchrom, d_rand_numbers);
		statistics_old();
	}

	void initmalloc()
	{
		rand_numbers = (int *) malloc(popsize);

		hipMalloc((void **)&d_gen,sizeof(int));
		hipMalloc((void **)&d_rand_numbers,sizeof(int));

		d_initmalloc<<<1,1>>>(d_popsize);
	}

	void updateGen()
	{
		hipMemcpy(d_gen, &gen, sizeof(int), hipMemcpyHostToDevice);
	}

	void generation()
	{

	}

	bestever sga(void) // la funzione principale da chiamare dopo aver settato tutti i parametri del
	{
		initmalloc();

		for(run=1; run<=maxruns; run++)
		{
			initialize();

			for(gen=0; gen<maxgen; gen++)
			{
				generation();
			}
		}

		return bestfit;
	}

}
